#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

#define TAM_MAP 30

//Kernel
__global__
void projecao_x(int *p_param){
	printf("video: %d\n",*p_param);	
	*p_param += 5; /*muda valor*/
}


int main(){
	int valor;
        valor = 10; /* declaração da variavel alocada no pc*/
	hipDeviceReset();
	
	int *d_param; /*Declaracao da variavel alocado na placa de video*/
	

	/*alocação de variaveis da placa de vídeos*/
	hipMalloc((void**)&d_param,sizeof(int));         /*aloca na cuda o tamanho de 1 inteiro*/

	
	hipMemcpy( d_param, &valor,
                              sizeof(int),
	                      hipMemcpyHostToDevice);

	printf("Chamando hello!\n");
	projecao_x << <1, 1>> >(d_param); /*Criação funcao do processador*/
	hipDeviceSynchronize();
	hipMemcpy( &valor, d_param,
                             sizeof(int),
                             hipMemcpyDeviceToHost);
	printf("pc:%d\n",valor);
	hipFree(d_param);
	return 0;
}
